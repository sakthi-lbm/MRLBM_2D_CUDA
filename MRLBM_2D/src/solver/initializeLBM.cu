#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>

#include "initializeLBM.cuh"

__global__ void gpu_initialize_moments(nodeVar fMom)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockIdx.y + threadIdx.y;

    dfloat rho = RHO_0;
    dfloat ux = toDFloat(0.0);
    dfloat uy = toDFloat(0.0);

    fMom.rho[IDX(x, y)] = rho - RHO_0;
    fMom.ux[IDX(x, y)] = ux;
    fMom.uy[IDX(x, y)] = uy;

    dfloat feq[Q];
    for (size_t i = 0; i < Q; i++)
    {
        dfloat Hxx = d_cx[i] * d_cx[i] - cs2;
        dfloat Hyy = d_cy[i] * d_cy[i] - cs2;
        dfloat Hxy = d_cx[i] * d_cy[i];

        feq[i] = w[i] * rho * (toDFloat(1.0) + as2 * (ux * d_cx[i] + uy * d_cy[i]) + toDFloat(0.5) * as2 * as2 * (Hxx * ux * ux + Hyy * uy * uy + toDFloat(2.0) * Hxy * ux * uy));
    }
}

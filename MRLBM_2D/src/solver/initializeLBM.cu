#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>

#include "initializeLBM.cuh"

__global__ void gpu_initialize_moments(nodeVar fMom)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockIdx.y + threadIdx.y;

    dfloat rho = RHO_0;
    dfloat ux = toDFloat(0.0);
    dfloat uy = toDFloat(0.0);

    fMom.rho[IDX(x, y)] = rho - RHO_0;
    fMom.ux[IDX(x, y)] = ux;
    fMom.uy[IDX(x, y)] = uy;

    dfloat pop_eq[Q];
    for (size_t i = 0; i < Q; i++)
    {
        dfloat Hxx = d_cx[i] * d_cx[i] - cs2;
        dfloat Hyy = d_cy[i] * d_cy[i] - cs2;
        dfloat Hxy = d_cx[i] * d_cy[i];

        pop_eq[i] = w[i] * rho * (toDFloat(1.0) + as2 * (ux * d_cx[i] + uy * d_cy[i]) + toDFloat(0.5) * as2 * as2 * (Hxx * ux * ux + Hyy * uy * uy + toDFloat(2.0) * Hxy * ux * uy));
    }
    dfloat inv_rho = toDFloat(1.0) / rho;
    fMom.mxx[IDX(x, y)] = (pop_eq[1] + pop_eq[3] + pop_eq[5] + pop_eq[6] + pop_eq[7] + pop_eq[8]) * inv_rho - cs2;
    fMom.myy[IDX(x, y)] = (pop_eq[2] + pop_eq[4] + pop_eq[5] + pop_eq[6] + pop_eq[7] + pop_eq[8]) * inv_rho - cs2;
    fMom.mxy[IDX(x, y)] = (pop_eq[5] - pop_eq[6] + pop_eq[7] - pop_eq[8]) * inv_rho;
}
